#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

#include <cutil.h>

#include "Convolution.h"

texture<float, 2, hipReadModeElementType> deviceMatrixTexture;
__device__ __constant__ float deviceXGradientMask[9];
__device__ __constant__ float deviceYGradientMask[9];
__device__ __constant__ float deviceGaussianFilterMask[25];

__global__ void deviceXGradientConvolution(float* output, unsigned matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;

	float accumulator = 0.0;

#pragma unroll
	for(int i = -1; i <= 1; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -1; j <= 1; ++j)
		{
			accumulator += deviceXGradientMask[(1 + i)* 3 + (1 + j)] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}

	output[outputRow * matrixWidth + outputColumn] = accumulator;
}

__global__ void deviceYGradientConvolution(float* output, unsigned matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
		int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;

	float accumulator = 0.0;

#pragma unroll
	for(int i = -1; i <= 1; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -1; j <= 1; ++j)
		{
			accumulator += deviceYGradientMask[(1 + i)* 3 + (1 + j)] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}

	output[outputRow * matrixWidth + outputColumn] = accumulator;
}

__global__ void deviceGaussianConvolution(float* output, unsigned matrixWidth)
{
	int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outputColumn = blockIdx.x * blockDim.x + threadIdx.x;
 
	float accumulator = 0.0;

#pragma unroll
	for(int i = -2; i <= 2; ++i)
	{
		unsigned matrixColumn = outputColumn + i;
#pragma unroll
		for(int j = -2; j <= 2; ++j)
		{
			accumulator += deviceGaussianFilterMask[(2 + i)* 3 + (2 + j)] * tex2D(deviceMatrixTexture, matrixColumn, outputRow + j);
		}
	}
	
	output[outputRow * matrixWidth + outputColumn] = accumulator / 159;
}

void initializeDevice()
{
	unsigned gradientMaskSize = 9 * sizeof(float);
	unsigned gaussianMaskSize = 25 * sizeof(float);

	// Copy kernels to device.
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceXGradientMask), xGradientMask, gradientMaskSize, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceYGradientMask), yGradientMask, gradientMaskSize, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceGaussianFilterMask), gaussianMask, gaussianMaskSize, 0, hipMemcpyHostToDevice));
}

void performConvolutionGpu(const float* inputMatrix, int matrixWidth, float* outputMatrix, ConvolutionType type)
{
	// Create timer.
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));

	// Compute memory sizes.
	unsigned matrixMemorySize = matrixWidth * matrixWidth * sizeof(float);
	
	// Set up device arrays.
	hipArray* deviceMatrixArray = NULL;
	float* deviceOutputArray = NULL;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMallocArray(&deviceMatrixArray, &channelDesc, matrixWidth, matrixWidth);
	CUDA_SAFE_CALL(hipMalloc((void**)&deviceOutputArray, matrixMemorySize));

	// Copy inputs to device.
	CUDA_SAFE_CALL(hipMemcpyToArray(deviceMatrixArray, 0, 0, inputMatrix, matrixMemorySize, hipMemcpyHostToDevice));

	// Set up image matrix as a texture.
	deviceMatrixTexture.addressMode[0] = hipAddressModeClamp;
	deviceMatrixTexture.addressMode[1] = hipAddressModeClamp;
	hipBindTextureToArray(deviceMatrixTexture, deviceMatrixArray);

	// Start timer.
	CUT_SAFE_CALL(cutStartTimer(timer));

	// Do it!
	dim3 dimGrid(matrixWidth / 16, matrixWidth / 16);
	dim3 dimBlock(16, 16);
	switch(type)
	{
		case GAUSSIAN:
			deviceGaussianConvolution<<<dimGrid, dimBlock>>>(deviceOutputArray, matrixWidth);
			break;
		case X_GRADIENT:
			deviceXGradientConvolution<<<dimGrid, dimBlock>>>(deviceOutputArray, matrixWidth);
			break;
		case Y_GRADIENT:
			deviceYGradientConvolution<<<dimGrid, dimBlock>>>(deviceOutputArray, matrixWidth);
			break;
	}

	// Check for errors.
	CUT_CHECK_ERROR("Kernel execution failed!");

	// Copy device result to host.
	CUDA_SAFE_CALL(hipMemcpy(outputMatrix, deviceOutputArray, matrixMemorySize, hipMemcpyDeviceToHost));

	// Stop and destroy timer, print results.
    CUT_SAFE_CALL(cutStopTimer(timer));
    //printf("Processing time for %dx%d matrix: %f ms\n", matrixWidth, matrixWidth, cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

	CUDA_SAFE_CALL(hipFreeArray(deviceMatrixArray));
	CUDA_SAFE_CALL(hipFree(deviceOutputArray));
	CUDA_SAFE_CALL(hipUnbindTexture(deviceMatrixTexture));
}